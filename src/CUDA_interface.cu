#include "hip/hip_runtime.h"
#include "CUDA_interface.h"

#if CERTFHE_USE_CUDA

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

namespace certFHE {

	const int CUDA_interface::MAX_THREADS_PER_BLOCK = 256;

	/****************** GPU KERNEL FUNCTIONS ******************/

	/**
	 * Device function
	 * Each thread operates on default length chunks
	**/
	__global__ static void ctxt_multiply_kernel(uint64_t deflen_to_uint64, uint64_t result_deflen_cnt, uint64_t snd_deflen_cnt,
		uint64_t * result, const uint64_t * fst, const uint64_t * snd) {

		int result_deflen_offset = blockDim.x * blockIdx.x + threadIdx.x;
		int result_deflen_stride = blockDim.x * gridDim.x;

		for (int result_deflen_i = result_deflen_offset; result_deflen_i < result_deflen_cnt; result_deflen_i += result_deflen_stride) {

			int fst_deflen_i = (result_deflen_i / snd_deflen_cnt) * deflen_to_uint64;
			int snd_deflen_i = (result_deflen_i % snd_deflen_cnt) * deflen_to_uint64;

			for (int i = 0; i < deflen_to_uint64; i++)
				result[i + result_deflen_i * deflen_to_uint64] = fst[i + fst_deflen_i] & snd[i + snd_deflen_i];
		}
	}

	/**
	 * Device function
	 * Each thread operates on default length chunks
	**/
	__global__ static void ctxt_decrypt_kernel(uint64_t deflen_to_uint64, uint64_t to_decrypt_deflen_cnt, const uint64_t * to_decrypt, const uint64_t * sk_mask,
		int * decryption_result) {

		int to_decrypt_deflen_offset = blockDim.x * blockIdx.x + threadIdx.x;
		int to_decrypt_deflen_stride = blockDim.x * gridDim.x;

		int local_decryption_result = 1;

		for (int to_decrypt_deflen_i = to_decrypt_deflen_offset; to_decrypt_deflen_i < to_decrypt_deflen_cnt; to_decrypt_deflen_i += to_decrypt_deflen_stride) {

			for (int i = 0; i < deflen_to_uint64; i++)
				local_decryption_result &= ((to_decrypt[to_decrypt_deflen_i * deflen_to_uint64 + i] & sk_mask[i]) ^ sk_mask[i]) == (uint64_t)0;

			(void)atomicXor(decryption_result, local_decryption_result);
		}
	}

	/**
	 * Device function
	 * Each thread operates on default length chunks
	**/
	__global__ static void ctxt_permute_kernel(uint64_t deflen_to_uint64, uint64_t to_permute_deflen_cnt, uint64_t * to_permute, const PermInversion * perm_inversions, uint64_t inv_cnt) {

		int to_permute_deflen_offset = blockDim.x * blockIdx.x + threadIdx.x;
		int to_permute_deflen_stride = blockDim.x * gridDim.x;

		int local_decryption_result = 1;

		for (int to_permute_deflen_i = to_permute_deflen_offset; to_permute_deflen_i < to_permute_deflen_cnt; to_permute_deflen_i += to_permute_deflen_stride) {

			uint64_t * current_chunk = to_permute + to_permute_deflen_i * deflen_to_uint64;
			uint64_t * current_chunk_res = to_permute + to_permute_deflen_i * deflen_to_uint64;

			for (int i = 0; i < inv_cnt; i++) {

				uint64_t fst_u64_ch = perm_inversions[i].fst_u64_ch;
				uint64_t snd_u64_ch = perm_inversions[i].snd_u64_ch;
				uint64_t fst_u64_r = perm_inversions[i].fst_u64_r;
				uint64_t snd_u64_r = perm_inversions[i].snd_u64_r;

				unsigned char val_i = (current_chunk[fst_u64_ch] >> fst_u64_r) & 0x01;
				unsigned char val_j = (current_chunk[snd_u64_ch] >> snd_u64_r) & 0x01;

				if (val_i)
					current_chunk_res[snd_u64_ch] |= (uint64_t)1 << snd_u64_r;
				else
					current_chunk_res[snd_u64_ch] &= ~((uint64_t)1 << snd_u64_r);

				if (val_j)
					current_chunk_res[fst_u64_ch] |= (uint64_t)1 << fst_u64_r;
				else
					current_chunk_res[fst_u64_ch] &= ~((uint64_t)1 << fst_u64_r);
			}
		}
	}

	/****************** CUDA INTERFACE METHODS ******************/

	__host__ void * CUDA_interface::RAM_TO_VRAM_copy(void * ram_address, uint64_t size_to_copy, void * vram_address) {

		if (!vram_address)
			hipMalloc(&vram_address, size_to_copy);

		hipMemcpy(vram_address, ram_address, size_to_copy, hipMemcpyHostToDevice);
		hipDeviceSynchronize();

		return vram_address;
	}

	__host__ void * CUDA_interface::VRAM_TO_RAM_copy(void * vram_address, uint64_t size_to_copy, void * ram_address) {

		if (!ram_address)
			ram_address = new char[size_to_copy];

		hipMemcpy(ram_address, vram_address, size_to_copy, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		return ram_address;
	}

	__host__ void * CUDA_interface::VRAM_TO_VRAM_copy(void * vram_address, uint64_t size_to_copy, void * vram_new_address) {

		if (!vram_new_address)
			hipMalloc(&vram_new_address, size_to_copy);

		hipMemcpy(vram_new_address, vram_address, size_to_copy, hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();

		return vram_new_address;
	}

	__host__ void CUDA_interface::VRAM_delete(void * vram_address) { hipFree(vram_address); }

	__host__ uint64_t * CUDA_interface::VRAM_VRAM_VRAM_chiphertext_multiply(uint64_t deflen_to_uint64, uint64_t fst_deflen_cnt, uint64_t snd_deflen_cnt,
		const uint64_t * fst, const uint64_t * snd) {

		uint64_t result_deflen_cnt = fst_deflen_cnt * snd_deflen_cnt;

		uint64_t * vram_result;
		hipMalloc(&vram_result, (uint64_t)result_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));

		int threads_per_block = result_deflen_cnt > MAX_THREADS_PER_BLOCK ? MAX_THREADS_PER_BLOCK : (int)result_deflen_cnt;

		int block_cnt = (int)(result_deflen_cnt / MAX_THREADS_PER_BLOCK);
		if (result_deflen_cnt % MAX_THREADS_PER_BLOCK)
			block_cnt += 1;

		ctxt_multiply_kernel << < block_cnt, threads_per_block >> > (deflen_to_uint64, result_deflen_cnt, snd_deflen_cnt, vram_result, fst, snd);
		hipDeviceSynchronize();

		return vram_result;
	}

	__host__ uint64_t * CUDA_interface::RAM_VRAM_VRAM_chiphertext_multiply(uint64_t deflen_to_uint64, uint64_t fst_deflen_cnt, uint64_t snd_deflen_cnt,
		const uint64_t * fst, const uint64_t * snd) {

		uint64_t * vram_fst = (uint64_t *)CUDA_interface::RAM_TO_VRAM_copy((void *)fst, fst_deflen_cnt  * deflen_to_uint64 * sizeof(uint64_t), 0);
		uint64_t * mul_result = CUDA_interface::VRAM_VRAM_VRAM_chiphertext_multiply(deflen_to_uint64, fst_deflen_cnt, snd_deflen_cnt, vram_fst, snd);

		hipFree(vram_fst);

		return mul_result;
	}

	__host__ uint64_t * CUDA_interface::RAM_RAM_VRAM_chiphertext_multiply(uint64_t deflen_to_uint64, uint64_t fst_deflen_cnt, uint64_t snd_deflen_cnt,
		const uint64_t * fst, const uint64_t * snd) {

		uint64_t * vram_fst = (uint64_t *)CUDA_interface::RAM_TO_VRAM_copy((uint64_t *)fst, fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), 0);
		uint64_t * vram_snd = (uint64_t *)CUDA_interface::RAM_TO_VRAM_copy((uint64_t *)snd, snd_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), 0);

		uint64_t * mul_result = CUDA_interface::VRAM_VRAM_VRAM_chiphertext_multiply(deflen_to_uint64, fst_deflen_cnt, snd_deflen_cnt, vram_fst, vram_snd);

		hipFree(vram_fst);
		hipFree(vram_snd);

		return mul_result;
	}

	__host__ uint64_t * CUDA_interface::VRAM_VRAM_VRAM_chiphertext_addition(uint64_t deflen_to_uint64, uint64_t fst_deflen_cnt, uint64_t snd_deflen_cnt,
		const uint64_t * fst, const uint64_t * snd) {

		uint64_t * add_result;
		hipMalloc(&add_result, (fst_deflen_cnt + snd_deflen_cnt) * deflen_to_uint64 * sizeof(uint64_t));

		CUDA_interface::VRAM_TO_VRAM_copy((void *)fst, fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), add_result);
		CUDA_interface::VRAM_TO_VRAM_copy((void *)snd, snd_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), ((char *)add_result) + fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));

		return add_result;
	}

	__host__ uint64_t * CUDA_interface::RAM_VRAM_VRAM_chiphertext_addition(uint64_t deflen_to_uint64, uint64_t fst_deflen_cnt, uint64_t snd_deflen_cnt,
		const uint64_t * fst, const uint64_t * snd) {

		uint64_t * add_result;
		hipMalloc(&add_result, (fst_deflen_cnt + snd_deflen_cnt) * deflen_to_uint64 * sizeof(uint64_t));

		CUDA_interface::RAM_TO_VRAM_copy((void *)fst, fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), add_result);
		CUDA_interface::VRAM_TO_VRAM_copy((void *)snd, snd_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), ((char *)add_result) + fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));

		return add_result;
	}

	__host__ uint64_t * CUDA_interface::RAM_RAM_VRAM_chiphertext_addition(uint64_t deflen_to_uint64, uint64_t fst_deflen_cnt, uint64_t snd_deflen_cnt,
		const uint64_t * fst, const uint64_t * snd) {

		uint64_t * add_result;
		hipMalloc(&add_result, (fst_deflen_cnt + snd_deflen_cnt) * deflen_to_uint64 * sizeof(uint64_t));

		CUDA_interface::RAM_TO_VRAM_copy((void *)fst, fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), add_result);
		CUDA_interface::RAM_TO_VRAM_copy((void *)snd, snd_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t), ((char *)add_result) + fst_deflen_cnt * deflen_to_uint64 * sizeof(uint64_t));

		return add_result;
	}

	__host__ int CUDA_interface::VRAM_ciphertext_decryption(uint64_t deflen_to_uint64, uint64_t to_decrypt_deflen_cnt, const uint64_t * to_decrypt, const uint64_t * sk_mask) {

		int * vram_decryption_result;

		hipMalloc(&vram_decryption_result, sizeof(int));
		hipMemset(vram_decryption_result, 0, sizeof(int));

		hipDeviceSynchronize();

		int threads_per_block = to_decrypt_deflen_cnt > MAX_THREADS_PER_BLOCK ? MAX_THREADS_PER_BLOCK : (int)to_decrypt_deflen_cnt;

		int block_cnt = (int)(to_decrypt_deflen_cnt / MAX_THREADS_PER_BLOCK);
		if (to_decrypt_deflen_cnt % MAX_THREADS_PER_BLOCK)
			block_cnt += 1;

		ctxt_decrypt_kernel <<< block_cnt, threads_per_block >>> (deflen_to_uint64, to_decrypt_deflen_cnt, to_decrypt, sk_mask, vram_decryption_result);
		hipDeviceSynchronize();

		int decryption_result;

		hipMemcpy(&decryption_result, vram_decryption_result, sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		hipFree(vram_decryption_result);

		return decryption_result;
	}

	__host__ void CUDA_interface::VRAM_ciphertext_permutation(uint64_t deflen_to_uint64, uint64_t to_permute_deflen_cnt, uint64_t * to_permute, 
																const PermInversion * perm_inversions, uint64_t inv_cnt) {
	
		int threads_per_block = to_permute_deflen_cnt > MAX_THREADS_PER_BLOCK ? MAX_THREADS_PER_BLOCK : (int)to_permute_deflen_cnt;

		int block_cnt = (int)(to_permute_deflen_cnt / MAX_THREADS_PER_BLOCK);
		if (to_permute_deflen_cnt % MAX_THREADS_PER_BLOCK)
			block_cnt += 1;

		ctxt_permute_kernel <<< block_cnt, threads_per_block >>> (deflen_to_uint64, to_permute_deflen_cnt, to_permute, perm_inversions, inv_cnt);
		hipDeviceSynchronize();
	}
}

#endif